#ifndef EDGEDATATYPE_CU
#define EDGEDATATYPE_CU

#include "../Algorithm/EdgeDataType.h"
#include "../MultipleGPU/MultiGraphStorage.h"
#include "../MedusaRT/Utilities.h"
#include "Configuration.h"



EdgeArray::EdgeArray()
{
	srcVertexID = NULL;
	dstVertexID = NULL;
#ifdef MESSAGING
	msgDstID = NULL;
	incoming_msg_flag = NULL;
#endif
	edgeOffset = NULL;
	level_count = 0;
	size = 0;
}

void EdgeArray::resize(int num)
{
	if(size != 0)
	{
		free(srcVertexID);
		free(dstVertexID);
#ifdef MESSAGING
		free(msgDstID);
		free(incoming_msg_flag);
#endif
	}
	size = num;
	CPUMalloc((void**)&srcVertexID,sizeof(int)*num);
	CPUMalloc((void**)&dstVertexID,sizeof(int)*num);
#ifdef MESSAGING
	CPUMalloc((void**)&msgDstID,sizeof(int)*num);
	CPUMalloc((void**)&incoming_msg_flag,sizeof(unsigned int)*num);
#endif
}

void EdgeArray::assign(int i, Edge e)
{
	srcVertexID[i] = e.srcVertexID;
	dstVertexID[i] = e.dstVertexID;
	//	msgDstID[i] = e.msgDstID;
}

void EdgeArray::buildMEG(GraphIR &graph)
{

#ifndef MEG
	//caution
	printf("The configuration doesn't indicate using MEG\n");
#else
	/* check if the graph is sorted in descending order*/
	for(int i = 0; i < graph.vertexNum - 1; i ++)
		if(graph.vertexArray[i].vertex.edge_count < graph.vertexArray[i+1].vertex.edge_count)
		{
			printf("Before convert to MEG, the GraphIR must be sorted\n");
			exit(-1);
		}


		/* count the number of edges in each level */
		level_count = 0;
		for(int i = 0; i <  graph.vertexNum; i ++)
			if(graph.vertexArray[i].vertex.edge_count > level_count)
				level_count = graph.vertexArray[i].vertex.edge_count;
		CPUMalloc((void**)&edgeOffset,sizeof(int)*level_count);
		memset(edgeOffset, 0, sizeof(int)*level_count);
		for(int i = 0; i < level_count; i ++)
		{
			for(int j = 0; j <  graph.vertexNum; j ++)
			{
				if(graph.vertexArray[j].vertex.edge_count > i)
					edgeOffset[i] ++;
			}
		}

		/* construct MEG from GraphIR */
		// construct edge array

		resize(graph.totalEdgeCount);
		int placeIndex;
		int levelIndex;
		for(int i = 0; i < graph.vertexNum; i ++)
		{
			placeIndex = i;
			levelIndex = 0;
			// loop through the edge list
			if(graph.vertexArray[i].vertex.edge_count)
			{
				EdgeNode *tempEdgeNode = graph.vertexArray[i].firstEdge;
				while(tempEdgeNode != NULL)
				{
					assign(placeIndex, tempEdgeNode->edge);
					tempEdgeNode = tempEdgeNode->nextEdge;
					placeIndex += edgeOffset[levelIndex ++];
				}
			}
		}

#ifdef MESSAGING

		//compute incoming message flag for the combiner
		int *vertex_edge_count = (int*)malloc(sizeof(int)*graph.vertexNum);
		for(int i = 0; i < graph.vertexNum; i ++)
			vertex_edge_count[i] = graph.vertexArray[i].incoming_edge_count;



		//compute prefix sum
		int last_edge_count = vertex_edge_count[0];
		vertex_edge_count[0] = 0;
		for(int i = 1; i < graph.vertexNum; i ++)
		{
			int temp_edge_count = vertex_edge_count[i];
			vertex_edge_count[i] = vertex_edge_count[i - 1] + last_edge_count;
			last_edge_count = temp_edge_count; 
		}

		memset(incoming_msg_flag, 0, sizeof(unsigned int)*size);
		for(int i = 0; i < graph.vertexNum; i ++)
		{
			incoming_msg_flag[vertex_edge_count[i]] = 1;
			//	printf("set %d to 1\n",vertex_edge_count[i]);
		}
		//compute reverse edge ID
		for(int i = 0; i < size; i ++)
			msgDstID[i] = vertex_edge_count[dstVertexID[i]] ++;

		free(vertex_edge_count);
#endif
#endif
}

void EdgeArray::buildAA(GraphIR &graph)
{
	/* construct MEG from GraphIR */
	// construct edge array
	//��ΪAAҪ����CPU�����ݽṹ����������Ҫ������������ʾ
#ifndef AA
	printf("The configuration doesn't indicate using AA\n");
	//exit(-1);
#endif
	resize(graph.totalEdgeCount);
	int placeIndex = 0;
	for(int i = 0; i < graph.vertexNum; i ++)
	{
		EdgeNode *tempEdgeNode = graph.vertexArray[i].firstEdge;
		
		while(tempEdgeNode != NULL)
		{
			
			//printf("%d - > %d\n",tempEdgeNode->edge.srcVertexID, tempEdgeNode->edge.dstVertexID);
			if(tempEdgeNode->edge.srcVertexID != i)
			{
				printf("edge src %d doesnot mat vertex ID %d\n",tempEdgeNode->edge.srcVertexID, i);
				exit(-1);
			}
			assign(placeIndex, tempEdgeNode->edge);
			tempEdgeNode = tempEdgeNode->nextEdge;
			placeIndex ++;
		}
	}
#ifdef MESSAGING	
	//compute incoming message flag for the combiner
	int *vertex_edge_count = (int*)malloc(sizeof(int)*graph.vertexNum);
	for(int i = 0; i < graph.vertexNum; i ++)
		vertex_edge_count[i] = graph.vertexArray[i].incoming_edge_count;



	//compute prefix sum
	int last_edge_count = vertex_edge_count[0];
	vertex_edge_count[0] = 0;
	for(int i = 1; i < graph.vertexNum; i ++)
	{
		int temp_edge_count = vertex_edge_count[i];
		vertex_edge_count[i] = vertex_edge_count[i - 1] + last_edge_count;
		last_edge_count = temp_edge_count; 
	}


	memset(incoming_msg_flag, 0, sizeof(unsigned int)*size);
	for(int i = 0; i < graph.vertexNum; i ++)
		incoming_msg_flag[vertex_edge_count[i]] = 1;
	//compute reverse edge ID
	for(int i = 0; i < size; i ++)
		msgDstID[i] = vertex_edge_count[dstVertexID[i]] ++;

	free(vertex_edge_count);
#endif


}






void EdgeArray::buildELL(GraphIR &graph)
{
	//find the maximum degree
#ifndef ELL
	//caution
	printf("The configuration doesn't indicate using ELL\n");

#else
	int max_degree = 0;
	for(int i = 0; i < graph.vertexNum; i ++)
		if(graph.vertexArray[i].vertex.edge_count > max_degree)
			max_degree = graph.vertexArray[i].vertex.edge_count;
	printf("ELL edge memory space %d\n", graph.vertexNum*max_degree); 
	resize(graph.vertexNum*max_degree);
	for(int i = 0; i < graph.vertexNum; i ++)
	{
		int placeIndex = i;
		EdgeNode *tempEdgeNode = graph.vertexArray[i].firstEdge;
		while(tempEdgeNode != NULL)
		{
			if(placeIndex > graph.vertexNum*max_degree)
				printf("placeIndex error\n");
			assign(placeIndex, tempEdgeNode->edge);
			tempEdgeNode = tempEdgeNode->nextEdge;
			placeIndex += graph.vertexNum;
		}
	}
#endif

}

/**
* The first part of the edge_array is ELL, the second part is AA
*
* @param   - 
* @return	
* @note	
*
*/
void EdgeArray::buildHY(GraphIR &graph, VertexArray &varr, int threshold)
{
	//find the length of AA

#ifndef HY
	//caution
	printf("The configuration doesn't indicate using HY\n");
#else

	int AA_length = graph.vertexNum*threshold;
	for(int i = 0; i < graph.vertexNum; i ++)
	{
		varr.edge_index[i] = AA_length;
		if(graph.vertexArray[i].vertex.edge_count > threshold)
		{
			AA_length += (graph.vertexArray[i].vertex.edge_count - threshold);
			varr.edge_count[i] = threshold;
		}
		else
			varr.edge_count[i] = graph.vertexArray[i].vertex.edge_count;

	}
	varr.edge_index[graph.vertexNum] = AA_length;
	printf("HY edge memory space %d index space %d\n", AA_length, graph.vertexNum);
	resize(AA_length);
	//	printf("AA_length + graph.vertexNum*threshold = %d",AA_length + graph.vertexNum*threshold);
	int ELL_count;
	int placeIndex;
	int AA_index = graph.vertexNum*threshold;
	for(int i = 0; i < graph.vertexNum; i ++)
	{
		placeIndex = i;
		ELL_count = 0;
		EdgeNode *tempEdgeNode = graph.vertexArray[i].firstEdge;
		while(tempEdgeNode != NULL)
		{
			if(ELL_count < threshold)
			{
				if(placeIndex >= AA_length + graph.vertexNum*threshold)
					printf("placeIndex error\n");
				assign(placeIndex, tempEdgeNode->edge);
				ELL_count ++;

			}
			else
				assign(AA_index ++, tempEdgeNode->edge);

			tempEdgeNode = tempEdgeNode->nextEdge;
			placeIndex += graph.vertexNum;
		}
	}
#endif	

}

















void D_EdgeArray::Fill(EdgeArray &ea)
{
	if(size != 0)
	{
		CUDA_SAFE_CALL(hipFree(d_srcVertexID));
		CUDA_SAFE_CALL(hipFree(d_dstVertexID));
#ifdef MESSAGING
		CUDA_SAFE_CALL(hipFree(d_msgDstID));
		CUDA_SAFE_CALL(hipFree(d_incoming_msg_flag));
#endif
#ifdef MEG
		CUDA_SAFE_CALL(hipFree(d_edgeOffset));
#endif
	}
	size = ea.size;
	GPUMalloc((void**)&d_srcVertexID,sizeof(int)*size);
	GPUMalloc((void**)&d_dstVertexID,sizeof(int)*size);
#ifdef MEG
	GPUMalloc((void**)&d_edgeOffset, sizeof(int)*ea.level_count);
	CUDA_SAFE_CALL(hipMemcpy(d_edgeOffset, ea.edgeOffset, sizeof(int)*ea.level_count, hipMemcpyHostToDevice));
#endif

	//	GPUMalloc((void**)&d_msgDstID,sizeof(int)*size);

	CUDA_SAFE_CALL(hipMemcpy(d_srcVertexID, ea.srcVertexID, sizeof(int)*size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_dstVertexID, ea.dstVertexID, sizeof(int)*size, hipMemcpyHostToDevice));
#ifdef MESSAGING	
	GPUMalloc((void**)&d_incoming_msg_flag,sizeof(unsigned int)*size);
	CUDA_SAFE_CALL(hipMemcpy(d_msgDstID, ea.msgDstID, sizeof(int)*size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_incoming_msg_flag, ea.incoming_msg_flag, sizeof(unsigned int)*size, hipMemcpyHostToDevice));
#endif

}

#endif
