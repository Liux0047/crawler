#include "../Algorithm/MessageDataType.h"
#include "../MedusaRT/Utilities.h"


//---------------------------------------------------------------------------------------------------------//

MessageArray::MessageArray()
{
	val = NULL;
	size = 0;
}

void MessageArray::resize(int new_size)
{
	if(size)
	{
		free(val);
	}
	size = new_size;
	CPUMalloc((void**)&val, sizeof(int)*new_size);

}

//---------------------------------------------------------------------------------------------------------//


void D_MessageArray::Fill(MessageArray ma)
{
	if(size)
	{
		CUDA_SAFE_CALL(hipFree(d_val));
	}
	size = ma.size;
	GPUMalloc((void**)&d_val, sizeof(MVT)*size);
	hipMemcpy(d_val, ma.val,sizeof(MVT)*size, hipMemcpyHostToDevice);
}

void D_MessageArray::resize(int new_size)
{
	if(size)
	{
		CUDA_SAFE_CALL(hipFree(d_val));
	}
	size = new_size;
	GPUMalloc((void**)&d_val, sizeof(MVT)*size);
}

